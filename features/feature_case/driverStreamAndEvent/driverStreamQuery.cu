#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipStream_t stream;

    // Initialize the CUDA Driver API
    hipInit(0);

    // Create a CUDA stream
    hipStreamCreateWithFlags(&stream, hipStreamDefault);

    hipError_t queryResult = hipStreamQuery(stream);

    if (queryResult == hipSuccess) {
        std::cout << "Kernel execution has completed." << std::endl;
    } else if (queryResult == hipErrorNotReady) {
        std::cout << "Kernel execution has not yet completed." << std::endl;
    } else {
        std::cerr << "Failed to query the stream status." << std::endl;
    }

    // Clean up resources
    hipStreamDestroy(stream);

    return 0;
}